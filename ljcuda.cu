#include "hip/hip_runtime.h"
#include "common.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

float *vector_alloc(int N)
{
    return (float *)malloc(N * sizeof(float));
}

__global__ void ode(
    int N,
    paramsStruct params,
    float *f,
    float *x)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int M = N / 4;
    for (int i = index; i < M * M; i += stride)
    {
        int j = 4 * (i / M);
        int k = 4 * (i % M);
        if (j == k)
        {
            f[j + R_X] = x[j + V_X];
            f[j + R_Y] = x[j + V_Y];
        }
        else
        {
            float r_kj_x = x[k + R_X] - x[j + R_X];
            float r_kj_y = x[k + R_Y] - x[j + R_Y];
            float r_kj_sq = r_kj_x * r_kj_x + r_kj_y * r_kj_y;
            if (r_kj_sq < params.r_z_sq)
            {
                float r_rel_sq = params.r_m_sq / r_kj_sq;
                float coeff = (params.b / r_kj_sq) * (r_rel_sq * r_rel_sq * r_rel_sq -
                                                      r_rel_sq * r_rel_sq * r_rel_sq * r_rel_sq * r_rel_sq * r_rel_sq);
                atomicAdd(&f[j + V_X], r_kj_x * coeff);
                atomicAdd(&f[j + V_Y], r_kj_y * coeff);
            }
        }
    }
}

__global__ void vector_reset(int N, float *k_1, float *k_2, float *k_3, float *k_4)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride)
    {
        k_1[i] = 0;
        k_2[i] = 0;
        k_3[i] = 0;
        k_4[i] = 0;
    }
}

__global__ void vector_add(int N, float *x, float *k, float mul, float *r)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride)
    {
        r[i] = x[i] + k[i] * mul;
    }
}

__global__ void vector_rk(int N, float dt, float *x, float *k_1, float *k_2, float *k_3, float *k_4)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int j = index; j < N; j += stride)
    {
        x[j] += (dt / 6) * (k_1[j] + 2 * k_2[j] + 2 * k_3[j] + k_4[j]);
    }
}

void compute_iteration(int N, int M_stride, float dt, paramsStruct params, float *x, float *k_1, float *k_2, float *k_3, float *k_4, float *x_tmp)
{
    int blockSize = 64;
    int numBlocksOde = min(512, N * N / blockSize + 1);
    int numBlocksVector = min(512, N / blockSize + 1);    
    for (int i = 0; i < M_stride; i++)
    {
        vector_reset<<<numBlocksVector, blockSize>>>(N, k_1, k_2, k_3, k_4);
        ode<<<numBlocksOde, blockSize>>>(N, params, k_1, x);
        vector_add<<<numBlocksVector, blockSize>>>(N, x, k_1, dt / 2, x_tmp);
        ode<<<numBlocksOde, blockSize>>>(N, params, k_2, x_tmp);
        vector_add<<<numBlocksVector, blockSize>>>(N, x, k_2, dt / 2, x_tmp);
        ode<<<numBlocksOde, blockSize>>>(N, params, k_3, x_tmp);
        vector_add<<<numBlocksVector, blockSize>>>(N, x, k_3, dt, x_tmp);
        ode<<<numBlocksOde, blockSize>>>(N, params, k_4, x_tmp);
        vector_rk<<<numBlocksVector, blockSize>>>(N, dt, x, k_1, k_2, k_3, k_4);
    }
}

void compute(
    int N,
    int M_out,
    int M_stride,
    float dt,
    paramsStruct params,
    float *x_0,
    float *x_out)
{
    float *x, *k_1, *k_2, *k_3, *k_4, *x_tmp;
    hipMalloc(&x, N * sizeof(float));
    hipMalloc(&k_1, N * sizeof(float));
    hipMalloc(&k_2, N * sizeof(float));
    hipMalloc(&k_3, N * sizeof(float));
    hipMalloc(&k_4, N * sizeof(float));
    hipMalloc(&x_tmp, N * sizeof(float));
    hipMemcpy(x, x_0, N * sizeof(float), hipMemcpyDefault);
    memcpy(x_out, x_0, N * sizeof(float));
    for (int i = 1; i < M_out; i++)
    {
        compute_iteration(N, M_stride, dt, params, x, k_1, k_2, k_3, k_4, x_tmp);
        float *x_out_ptr = x_out + i * N;
        hipMemcpy(x_out_ptr, x, N * sizeof(float), hipMemcpyDefault);
        printf("Step: %d/%d, Progress: %g\n", i * M_stride, (M_out - 1) * M_stride, 100.0 * i / (M_out - 1));
        energy(N, params, x_out_ptr);
    }
}

int main(int argc, const char *argv[])
{
    return entry_point(argc, argv, &compute);
}
